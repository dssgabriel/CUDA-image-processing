#include "config.hpp"
#include "driver.cuh"
#include "kernels.cuh"
#include "utils.hpp"

#include <cassert>

static constexpr size_t BLOCK_SIZE = 32;

namespace driver {
    auto run_on_target(
        Config const& cfg,
        FIBITMAP* bitmap,
        std::vector<uint8_t>& img,
        size_t width,
        size_t height,
        size_t pitch
    ) -> int32_t {
        int32_t ret = 0;

        switch (cfg.target) {
            case TargetKind::Host:
                ret = host::run(cfg, img, width, height, pitch);
                break;
            case TargetKind::Device:
                ret = device::run(cfg, bitmap, img, width, height, pitch);
                break;
        }

        return ret;
    }

    namespace host {
        auto run(
            Config const& cfg,
            std::vector<uint8_t>& img,
            size_t width,
            size_t height,
            size_t pitch
        ) -> int32_t {
            int32_t ret = 0;

            size_t size = 3 * width * height * sizeof(uint8_t);

            // Allocate memory for images on host
            auto h_img = (uint8_t*)(malloc(size));
            auto h_tmp = (uint8_t*)(malloc(size));

            // Copy original image on host-allocated images
            memcpy(h_img, img.data(), size);
            memcpy(h_tmp, img.data(), size);

            for (auto const& f: cfg.filters) {
                switch (f.kind) {
                    case FilterKind::Blur:
                        kernels::host::iterative_blur(
                            h_img,
                            width,
                            height,
                            f.associated_val.nb_iterations
                        );
                        break;
                    case FilterKind::Diapositive:
                        kernels::host::diapositive(h_img, width, height);
                        break;
                    case FilterKind::Grayscale:
                        kernels::host::grayscale(h_img, width, height);
                        break;
                    case FilterKind::HorizontalFlip:
                        kernels::host::flip_horizontally(h_img, h_tmp, width, height);
                        break;
                    case FilterKind::PopArt:
                        fprintf(
                            stderr,
                            "\033[1;33mwarning:\033[0m filter `Pop-Art` is not available on "
                            "host/CPU. Skipping...\n"
                        );
                        break;
                    case FilterKind::Saturate:
                        kernels::host::saturate_color(
                            h_img,
                            width,
                            height,
                            f.associated_val.color_to_saturate
                        );
                        break;
                    case FilterKind::Sobel:
                        kernels::host::grayscale(h_img, width, height);
                        memcpy(h_tmp, h_img, size);
                        kernels::host::sobel(
                            h_img,
                            h_tmp,
                            width,
                            height,
                            f.associated_val.threshold
                        );
                        break;
                }
            }

            memcpy(img.data(), h_img, size);

            free(h_img);
            free(h_tmp);

            return ret;
        }
    } // namespace host

    namespace device {
        auto run(
            Config const& cfg,
            FIBITMAP* bitmap,
            std::vector<uint8_t>& img,
            size_t width,
            size_t height,
            size_t pitch
        ) -> int32_t {
            hipError_t ret = hipSuccess;

            size_t size = 3 * width * height * sizeof(uint8_t);
            assert(img.size() == size);
            size_t width_split = width / 2;
            size_t height_split = height / 2;
            size_t size_split = 3 * width_split * height_split * sizeof(uint8_t);

            // Allocate memory for images on host
            uint8_t* h_img;
            hipHostMalloc((void**)(&h_img), size);
    
            // Allocate memory for images on device
            uint8_t* d_img;
            hipMalloc((void**)(&d_img), size);
    
            uint8_t* d_tmp;
            hipHostMalloc((void**)(&d_tmp), size);

            // Copy original image on host-allocated images
            memcpy(h_img, img.data(), size);

            // Copy original image on host-allocated images
            hipMemcpy(d_img, h_img, size, hipMemcpyHostToDevice);
            hipMemcpy(d_tmp, h_img, size, hipMemcpyHostToDevice);

            dim3 grid(width / BLOCK_SIZE + 1, height / BLOCK_SIZE + 1);
            dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            dim3 grid_popart(
                width_split / BLOCK_SIZE + 1,
                height_split / BLOCK_SIZE + 1
            );

            // Resize image for pop-art
            FIBITMAP* shrank_bitmap = FreeImage_Rescale(
                bitmap,
                width_split,
                height_split,
                FILTER_BOX
            );
            std::vector<uint8_t> tmp = utils::load_image(
                shrank_bitmap,
                FreeImage_GetWidth(shrank_bitmap),
                FreeImage_GetHeight(shrank_bitmap),
                FreeImage_GetPitch(shrank_bitmap)
            );

            for (auto const& f: cfg.filters) {
                switch (f.kind) {
                    case FilterKind::Blur:
                        for (size_t _ = 0; _ < f.associated_val.nb_iterations; ++_) {
                            kernels::device::blur<<<grid, block>>>(d_img, width, height);
                        }
                        break;
                    case FilterKind::Diapositive:
                        kernels::device::diapositive<<<grid, block>>>(d_img, width, height);
                        break;
                    case FilterKind::Grayscale:
                        kernels::device::grayscale<<<grid, block>>>(d_img, width, height);
                        break;
                    case FilterKind::HorizontalFlip:
                        kernels::device::flip_horizontally<<<grid, block>>>(
                            d_img,
                            d_tmp,
                            width,
                            height
                        );
                        break;
                    case FilterKind::PopArt:
                        // Allocate small images on host
                        uint8_t* h_img_tl;
                        hipHostMalloc((void**)(&h_img_tl), size_split);

                        uint8_t* h_img_tr;
                        hipHostMalloc((void**)(&h_img_tr), size_split);

                        uint8_t* h_img_bl;
                        hipHostMalloc((void**)(&h_img_bl), size_split);

                        uint8_t* h_img_br;
                        hipHostMalloc((void**)(&h_img_br), size_split);

                        // Allocate small images on device
                        uint8_t* d_img_tl;
                        hipMalloc((void**)(&d_img_tl), size_split);

                        uint8_t* d_img_tr;
                        hipMalloc((void**)(&d_img_tr), size_split);

                        uint8_t* d_img_bl;
                        hipMalloc((void**)(&d_img_bl), size_split);

                        uint8_t* d_img_br;
                        hipMalloc((void**)(&d_img_br), size_split);

                        // Create streams
                        hipStream_t stream[4];
                        for (size_t i = 0; i < 4; ++i) {
                            hipStreamCreate(&stream[i]);
                        }

                        // Load small images
                        memcpy(h_img_tl, tmp.data(), size_split);
                        memcpy(h_img_tr, h_img_tl, size_split);
                        memcpy(h_img_bl, h_img_tl, size_split);
                        memcpy(h_img_br, h_img_tl, size_split);

                        // Copy host-allocated small images on device
                        hipMemcpyAsync(
                            d_img_tl,
                            h_img_tl,
                            size_split,
                            hipMemcpyHostToDevice,
                            stream[0]
                        );
                        hipMemcpyAsync(
                            d_img_tr,
                            h_img_tr,
                            size_split,
                            hipMemcpyHostToDevice,
                            stream[1]
                        );
                        hipMemcpyAsync(
                            d_img_bl,
                            h_img_bl,
                            size_split,
                            hipMemcpyHostToDevice,
                            stream[2]
                        );
                        hipMemcpyAsync(
                            d_img_br,
                            h_img_br,
                            size_split,
                            hipMemcpyHostToDevice,
                            stream[3]
                        );

                        // Launch kernels
                        kernels::device::saturate_color<<<grid_popart, block, 0, stream[0]>>>(
                            d_img_tl,
                            width_split,
                            height_split,
                            Red
                        );
                        kernels::device::saturate_color<<<grid_popart, block, 0, stream[1]>>>(
                            d_img_tr,
                            width_split,
                            height_split,
                            Green
                        );
                        kernels::device::saturate_color<<<grid_popart, block, 0, stream[2]>>>(
                            d_img_bl,
                            width_split,
                            height_split,
                            Blue
                        );
                        kernels::device::grayscale<<<grid_popart, block, 0, stream[3]>>>(
                            d_img_br,
                            width_split,
                            height_split
                        );

                        // Copy small images from device to host
                        hipMemcpyAsync(
                            h_img_tl,
                            d_img_tl,
                            size_split,
                            hipMemcpyDeviceToHost,
                            stream[0]
                        );
                        hipMemcpyAsync(
                            h_img_tr,
                            d_img_tr,
                            size_split,
                            hipMemcpyDeviceToHost,
                            stream[1]
                        );
                        hipMemcpyAsync(
                            h_img_bl,
                            d_img_bl,
                            size_split,
                            hipMemcpyDeviceToHost,
                            stream[2]
                        );
                        hipMemcpyAsync(
                            h_img_br,
                            d_img_br,
                            size_split,
                            hipMemcpyDeviceToHost,
                            stream[3]
                        );
                        hipDeviceSynchronize();

                        for (size_t y = 0; y < height_split; y++) {
                            size_t idx_sub_img = y * width_split * 3;
                            size_t idx_img_ul = y * width * 3 + height_split * width * 3;
                            size_t idx_img_ur = y * width * 3 + height_split * width * 3 + width_split * 3;
                            size_t idx_img_dl = y * width * 3;
                            size_t idx_img_dr = y * width * 3 + width_split * 3;

                            utils::imgcpy(h_img, h_img_tl, width_split, idx_img_ul, idx_sub_img);
                            utils::imgcpy(h_img, h_img_tr, width_split, idx_img_ur, idx_sub_img);
                            utils::imgcpy(h_img, h_img_bl, width_split, idx_img_dl, idx_sub_img);
                            utils::imgcpy(h_img, h_img_br, width_split, idx_img_dr, idx_sub_img);
                        }

                        for (size_t i = 0; i < 4; ++i) {
                            hipStreamDestroy(stream[i]);
                        }

                        hipFree(d_img_tl);
                        hipFree(d_img_tr);
                        hipFree(d_img_bl);
                        hipFree(d_img_br);

                        hipHostFree(h_img_tl);
                        hipHostFree(h_img_tr);
                        hipHostFree(h_img_bl);
                        hipHostFree(h_img_br);

                        hipMemcpy(d_img, h_img, size, hipMemcpyHostToDevice);
                        break;
                    case FilterKind::Saturate:
                        kernels::device::saturate_color<<<grid, block>>>(
                            d_img,
                            width,
                            height,
                            f.associated_val.color_to_saturate
                        );
                        break;
                    case FilterKind::Sobel:
                        kernels::device::grayscale<<<grid, block>>>(d_img, width, height);
                        hipMemcpy(d_tmp, d_img, size, hipMemcpyDeviceToDevice);
                        kernels::device::sobel<<<grid, block>>>(
                            d_img,
                            d_tmp,
                            width,
                            height,
                            f.associated_val.threshold
                        );
                        break;
                }
            }

            hipMemcpy(img.data(), d_img, size, hipMemcpyDeviceToHost);

            hipHostFree(h_img);
            hipFree(d_img);
            hipFree(d_tmp);

            return ret;
        }
    } // namespace device
} // namespace driver
